

void allocateFeature(CvHaarFeature* d_f, CvHaarFeature* h_f)
{
    int size = sizeof(int) + 3*(sizeof(CvRect) + sizeof(float));
    //int size = sizeof(CvHaarFeature);
    hipMalloc((void**) &d_f, size);
    hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);
}

void allocateClassifier(CvHaarClassifier* d_c, CvHaarClassifier* h_c)
{
    int classifierSize = sizeof(CvHaarClassifier);
    hipMalloc((void**) &d_c, classifierSize);
    int* left, right;
    float* threshold, alpha;

    hipMalloc((void**) &threshold, sizeof(float));
    hipMalloc((void**) &alpha, sizeof(float));
    hipMemcpy(threshold, h_c->threshold, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(alpha, h_c->alpha, sizeof(float), hipMemcpyHostToDevice);

}